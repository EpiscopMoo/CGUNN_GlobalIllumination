#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_vector_types.h>
#include ""
#include "cutil_math.h"
#include "predefs.cuh"
#include "entities.cuh"
#include "path_tracing.cuh"

// SCENE
// { float radius, { float3 position }, { float3 emission }, { float3 colour }, refl_type }
__constant__ Sphere spheres[] = 
{
	{ 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.99f, 0.25f, 0.25f }, SPEC }, //����� ����� 
	{ 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .99f }, DIFF }, //������ 
	{ 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //������� 
	{ 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { .0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF }, //��������, �� � �� �� ����� �������� 
	{ 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, GENR }, //��� 
	{ 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //������� 
	{ 16.5f, { 18.0f, 49.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // ��� 1
	{ 16.5f, { 73.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 0.99f, 0.99f, 1.0f }, SPEC }, // ��� 2
	{ 16.5f, { 30.0f, 20.0f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 0.99f, 0.99f, 0.99f }, REFR }, // ��� 3
	{ 600.0f, { 50.0f, 681.6f - .77f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // ����
};

__global__ void render_kernel(float3 *output, Sphere* spheres, int count, float* presets, int preset_num, int samps)
{
	Preset preset(presets, preset_num);
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int i = (height - y - 1)*width + x; // index of current pixel (calculated using thread index) 

	unsigned int s1 = x;  // seeds for random number generator
	unsigned int s2 = y;

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1)));
	float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f);
	float3 cy = normalize(cross(cx, cam.dir)) * .5135;
	float3 r; // r is final pixel color       
    
	r = make_float3(0.0f);

	for (int s = 0; s < samps; s++)
	{    
		float3 d = cam.dir + cx*((.25 + x) / width - .5) + cy*((.25 + y) / height - .5);
  
		Ray cam_ray(cam.orig + d * 40, normalize(d));
		r = r + radiance(
			spheres, count, 
			cam_ray, &s1, &s2, &preset)*(1. / samps); 
	}
	output[i] = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
}

float3* cuda_main(int* w, int* h, float* cpu_presets, int preset_num, int samples = 128)
{
	*w = width;
	*h = height;
	float3* output_h = new float3[width*height]; // pointer to memory for image on the host (system RAM)
	float3* output_d;    // pointer to memory for image on the device (GPU VRAM)
	float* cuda_presets; //��� �� ��������� ���������� (GPU)

	// allocate memory on the CUDA device (GPU VRAM)
	hipMalloc(&output_d, width * height * sizeof(float3));
	hipMalloc(&cuda_presets, preset_num*sizeof(float));
	
	hipMemcpy(cuda_presets, cpu_presets, preset_num*sizeof(float), hipMemcpyHostToDevice);
        
	// dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 block(8, 8, 1);   
	dim3 grid(width / block.x, height / block.y, 1);
	
	// schedule threads on device and launch CUDA kernel from host
	render_kernel<<<grid, block>>>(output_d, spheres, 10, cuda_presets, preset_num, samples);  

	// copy results of computation from device back to host
	hipMemcpy(output_h, output_d, width * height *sizeof(float3), hipMemcpyDeviceToHost);  
 
	// free CUDA memory
	hipFree(cuda_presets);
	hipFree(output_d);
	return output_h;
}